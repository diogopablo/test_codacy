#include "hip/hip_runtime.h"
#include <iostream>  
#include <hipcub/hipcub.hpp> // Inclusion de la bibliothèque CUB  

__global__ void add(int* a, int* b, int* c, int size) {  
    int index = threadIdx.x;   
    if (index < size) {  
        c[index] = a[index] + b[index]; // Utilisation de pointeurs  
    }  
}  

int main() {  
    const int arraySize = 5;  
    int a[arraySize] = {1, 2, 3, 4, 5};  
    int b[arraySize] = {10, 20, 30, 40, 50};  
    int c[arraySize];  

    int *d_a, *d_b, *d_c;  
    hipMalloc(&d_a, arraySize * sizeof(int));  
    hipMalloc(&d_b, arraySize * sizeof(int));  
    hipMalloc(&d_c, arraySize * sizeof(int));  

    hipMemcpy(d_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);  
    hipMemcpy(d_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);  

    add<<<1, arraySize>>>(d_a, d_b, d_c);  
    hipMemcpy(c, d_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);  

    for (int i = 0; i < arraySize; i++) {  
        std::cout << c[i] << " ";  
    }  

    hipFree(d_a);  
    hipFree(d_b);  
    hipFree(d_c);  
    return 0;  
}